#include "hip/hip_runtime.h"
#include "wtk_cudnn_cfg.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "wtk_cudnn.h"
#include "hipblas.h"
#ifndef THREAD_PER_BLOCK
#define THREAD_PER_BLOCK 1024
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif


wtk_cudnn_gpu_cfg_t* wtk_cudnn_gpu_cfg_new(int idx)
{
	wtk_cudnn_gpu_cfg_t *cfg;

	cfg=(wtk_cudnn_gpu_cfg_t*)wtk_malloc(sizeof(wtk_cudnn_gpu_cfg_t));
	cfg->idx=idx;
	cfg->max_thread=0;
	cfg->nlayer=0;
	cfg->trans=NULL;
	cfg->layer=NULL;
	return cfg;
}

void wtk_cudnn_gpu_cfg_delete(wtk_cudnn_gpu_cfg_t *cfg)
{
	int i;

	if(cfg->trans)
	{
		wtk_cudnn_trans_delete_cuda(cfg->trans);
	}
	if(cfg->layer)
	{
		for(i=0;i<cfg->nlayer;++i)
		{
			wtk_cudnn_layer_delete_cuda(cfg->layer[i]);
		}
		wtk_free(cfg->layer);
	}
	wtk_free(cfg);
}


void print_float_cu(float *a,int n)
{
	float *v;

	v=(float*)wtk_malloc(n*sizeof(float));
	hipMemcpy(v,a,n*sizeof(float),hipMemcpyDeviceToHost);
	//print_float(v,m->row*m->col);
	print_float(v,n);
	wtk_free(v);
}

void wtk_cudnn_vector_delete_cuda(wtk_cudnn_vector_t *v)
{
	if(v->v)
	{
		hipFree(v->v);
	}
	wtk_free(v);
}

void wtk_cudnn_trans_delete_cuda(wtk_cudnn_trans_t *v)
{
	if(v->b)
	{
		wtk_cudnn_vector_delete_cuda(v->b);
	}
	if(v->w)
	{
		wtk_cudnn_vector_delete_cuda(v->w);
	}
	wtk_free(v);
}

void wtk_cudnn_matrix_delete_cuda(wtk_cudnn_matrix_t *m)
{
	if(m->v)
	{
		hipFree(m->v);
	}
	wtk_free(m);
}

void wtk_cuda_print_err()
{
	hipError_t err;
	const char *msg;

	err=hipGetLastError();
	msg=hipGetErrorString(err);
	wtk_debug("cuda[%d]:%s\n",err,msg);
}

wtk_cudnn_matrix_t* wtk_cudnn_matrix_new_cuda(int row,int col)
{
	wtk_cudnn_matrix_t *m;
	int ret;

	m=(wtk_cudnn_matrix_t*)wtk_malloc(sizeof(wtk_cudnn_matrix_t));
	m->row=row;
	m->col=col;
	ret=hipMalloc((void**)&(m->v),row*col*sizeof(float));
	if(ret!=0)
	{
		goto end;
	}
	ret=hipMemset((m->v),0,row*col*sizeof(float));
	if(ret!=0)
	{
		wtk_cuda_print_err();
		goto end;
	}
end:
	if(ret!=0)
	{
		wtk_free(m);
		m=NULL;
	}
	return m;
}


void wtk_cudnn_layer_delete_cuda(wtk_cudnn_layer_t *layer)
{
	if(layer->b)
	{
		wtk_cudnn_vector_delete_cuda(layer->b);
	}
	if(layer->w)
	{
		wtk_cudnn_matrix_delete_cuda(layer->w);
	}
	wtk_free(layer);
}

wtk_cudnn_vector_t* wtk_cudnn_vector_host_to_cuda(wtk_cudnn_vector_t *vh)
{
	wtk_cudnn_vector_t *vd;
	int ret;
	int n=sizeof(float)*vh->len;

	vd=(wtk_cudnn_vector_t*)wtk_malloc(sizeof(wtk_cudnn_vector_t));
	vd->len=vh->len;
	vd->v=NULL;
	ret=hipMalloc((void**)&(vd->v),n);
	if(ret!=0){goto end;}
	ret=hipMemcpy(vd->v,vh->v,n,hipMemcpyHostToDevice);
	if(ret!=0){goto end;}
end:
	if(ret!=0)
	{
		wtk_cudnn_vector_delete_cuda(vd);
		vd=NULL;
	}
	return vd;
}

int wtk_cudnn_cfg_update_prop(wtk_cudnn_cfg_t *cfg)
{
	hipDeviceProp_t prop;
	int i,cnt;
	int ret;

	ret=hipGetDeviceCount(&(cnt));
	if(ret!=0){goto end;}
	cfg->ngpu=cnt;
	cfg->gpu=(wtk_cudnn_gpu_cfg_t**)wtk_calloc(cnt,sizeof(wtk_cudnn_gpu_cfg_t*));
	for(i=0;i<cnt;++i)
	{
		ret=hipGetDeviceProperties(&(prop),i);
		if(ret!=0){goto end;}
		cfg->gpu[i]=wtk_cudnn_gpu_cfg_new(i);
		cfg->gpu[i]->max_thread=prop.maxThreadsPerBlock;
	}
end:
	return ret;
}

wtk_cudnn_trans_t* wtk_cudnn_trans_new_cuda(wtk_cudnn_trans_t *trans)
{
	wtk_cudnn_trans_t *vt;
	int ret=-1;

	vt=(wtk_cudnn_trans_t*)wtk_malloc(sizeof(wtk_cudnn_trans_t));
	vt->b=NULL;
	vt->w=NULL;
	vt->b=wtk_cudnn_vector_host_to_cuda(trans->b);
	if(!vt->b){goto end;}
	vt->w=wtk_cudnn_vector_host_to_cuda(trans->w);
	if(!vt->w){goto end;}
	ret=0;
end:
	if(ret!=0)
	{
		wtk_cudnn_trans_delete_cuda(vt);
		vt=NULL;
	}
	return vt;
}

wtk_cudnn_matrix_t* wtk_cudnn_matrix_host_to_cuda(wtk_cudnn_matrix_t *mh,int blk_size)
{
	wtk_cudnn_matrix_t *md;
	int ret;
	int n;

	md=(wtk_cudnn_matrix_t*)wtk_malloc(sizeof(wtk_cudnn_matrix_t));
	md->row=mh->row;
	md->col=mh->col;
	md->v=NULL;
	n=((mh->row-1)/blk_size+1)*blk_size;
	//wtk_debug("row=%d/%d\n",m->row,n);
	n=n*mh->col*sizeof(float);
	ret=hipMalloc((void**)&(md->v),n);
	if(ret!=0){goto end;}
	ret=hipMemset(md->v,0,n);
	if(ret!=0){goto end;}
	n=mh->row*mh->col*sizeof(float);
	ret=hipMemcpy(md->v,mh->v,n,hipMemcpyHostToDevice);
	if(ret!=0){goto end;}
end:
	if(ret!=0)
	{
		wtk_cudnn_matrix_delete_cuda(md);
		md=NULL;
	}
	return md;
}

wtk_cudnn_vector_t* wtk_cudnn_vector_host_to_cuda2(wtk_cudnn_vector_t *vh,int blk_size)
{
	wtk_cudnn_vector_t *vd;
	int ret;
	int n;

	vd=(wtk_cudnn_vector_t*)wtk_malloc(sizeof(wtk_cudnn_vector_t));
	vd->len=vh->len;
	n=((vh->len-1)/blk_size+1)*blk_size;
	//wtk_debug("n=%d len=%d\n",n,v->len);
	n=sizeof(float)*n;//v->len;
	ret=hipMalloc((void**)&(vd->v),n);
	if(ret!=0){goto end;}
	ret=hipMemset(vd->v,0,n);
	if(ret!=0){goto end;}
	ret=hipMemcpy(vd->v,vh->v,vh->len*sizeof(float),hipMemcpyHostToDevice);
	if(ret!=0){goto end;}
end:
	if(ret!=0)
	{
		wtk_cudnn_vector_delete_cuda(vd);
		vd=NULL;
	}
	return vd;
}


wtk_cudnn_layer_t* wtk_cudnn_layer_new_cuda(wtk_cudnn_layer_t *layer,int blk_size)
{
	wtk_cudnn_layer_t *ld;
	int ret;

	ld=(wtk_cudnn_layer_t*)wtk_malloc(sizeof(wtk_cudnn_layer_t));
	ld->type=layer->type;
	ld->b=NULL;
	ld->w=NULL;
	ld->b=wtk_cudnn_vector_host_to_cuda2(layer->b,blk_size);
	if(!ld->b){ret=-1;goto end;}
	ld->w=wtk_cudnn_matrix_host_to_cuda(layer->w,blk_size);
	if(!ld->w){ret=-1;goto end;}
	ret=0;
end:
	if(ret!=0)
	{
		wtk_cudnn_layer_delete_cuda(ld);
		ld=NULL;
	}
	return ld;
}


int wtk_cudnn_cfg_update_cuda(wtk_cudnn_cfg_t *cfg)
{
	wtk_cudnn_gpu_cfg_t *gpu;
	int ret;
	int i,j;

	ret=wtk_cudnn_cfg_update_prop(cfg);
	if(ret!=0){goto end;}
	if(cfg->blk_size>BLOCK_SIZE)
	{
		wtk_debug("blk_size %d/%d wrong.\n",cfg->blk_size,BLOCK_SIZE);
		ret=-1;
		goto end;
	}
	for(i=0;i<cfg->ngpu;++i)
	{
		hipSetDevice(i);
		gpu=cfg->gpu[i];
		gpu->trans=wtk_cudnn_trans_new_cuda(cfg->trans);
		if(!gpu->trans){ret=-1;goto end;}
		gpu->nlayer=cfg->nlayer;
		gpu->layer=(wtk_cudnn_layer_t**)wtk_malloc(sizeof(wtk_cudnn_layer_t*)*cfg->nlayer);
		for(j=0;j<cfg->nlayer;++j)
		{
			gpu->layer[j]=wtk_cudnn_layer_new_cuda(cfg->layer[j],cfg->blk_size);
			if(!gpu->layer[j]){ret=-1;goto end;}
		}
	}
	if(cfg->trans)
	{
		wtk_cudnn_trans_delete(cfg->trans);
		cfg->trans=NULL;
	}
	if(cfg->layer)
	{
		for(i=0;i<cfg->nlayer;++i)
		{
			wtk_cudnn_layer_delete(cfg->layer[i]);
		}
		wtk_free(cfg->layer);
		cfg->layer=NULL;
	}
	ret=0;
end:
	return ret;
}


#include "wtk_cudnn_env.h"

wtk_cudnn_matrix_t* wtk_cudnn_matrix_new_cuda2(int row,int col,int blk_size)
{
	wtk_cudnn_matrix_t *md;
	int ret;
	int n;

	md=(wtk_cudnn_matrix_t*)wtk_malloc(sizeof(wtk_cudnn_matrix_t));
	md->row=row;
	md->col=col;
	md->v=NULL;
	n=((row-1)/blk_size+1)*blk_size;
	//wtk_debug("row=%d/%d\n",m->row,n);
	n=n*col*sizeof(float);
	ret=hipMalloc((void**)&(md->v),n);
	if(ret!=0){goto end;}
	ret=hipMemset(md->v,0,n);
end:
	if(ret!=0)
	{
		wtk_cudnn_matrix_delete_cuda(md);
		md=NULL;
	}
	return md;
}


wtk_cudnn_env_t* wtk_cudnn_env_new(wtk_cudnn_cfg_t *cfg,wtk_cudnn_gpu_cfg_t *gpu_cfg)
{
	wtk_cudnn_env_t *env;
	int i;

	env=(wtk_cudnn_env_t*)wtk_malloc(sizeof(wtk_cudnn_env_t));
	env->cu_cfg=cfg;
	env->gpu_cfg=gpu_cfg;
	if(gpu_cfg)
	{
		hipSetDevice(gpu_cfg->idx);
	}
	env->output=(wtk_cudnn_matrix_t**)wtk_calloc(cfg->nlayer,sizeof(wtk_cudnn_matrix_t*));
	env->host_output=NULL;
	if(cfg->use_cuda)
	{
		env->input=wtk_cudnn_matrix_new_cuda(cfg->cache_size,gpu_cfg->trans->b->len);
		for(i=0;i<cfg->nlayer;++i)
		{
			//env->output[i]=wtk_cudnn_matrix_new_cuda2(cfg->cache_size,gpu_cfg->layer[i]->w->row,cfg->blk_size);
			env->output[i]=wtk_cudnn_matrix_new_cuda(cfg->cache_size,gpu_cfg->layer[i]->w->row);
			if(i==cfg->nlayer-1)
			{
				env->host_output=wtk_cudnn_matrix_new(cfg->cache_size,gpu_cfg->layer[i]->w->row);
			}
		}
	}else
	{
		env->input=NULL;//wtk_cudnn_matrix_new(cfg->cache_size,cfg->trans->b->len);
		for(i=0;i<cfg->nlayer;++i)
		{
			env->output[i]=wtk_cudnn_matrix_new(cfg->cache_size,cfg->layer[i]->w->row);//,cfg->layer[i]->w->col);
		}
	}
	return env;
}

void wtk_cudnn_env_delete(wtk_cudnn_env_t *env)
{
	int i;

	if(env->host_output)
	{
		wtk_cudnn_matrix_delete(env->host_output);
	}
	if(env->input)
	{
		wtk_cudnn_matrix_delete_cuda(env->input);
	}
	for(i=0;i<env->cu_cfg->nlayer;++i)
	{
		if(env->cu_cfg->use_cuda)
		{
			wtk_cudnn_matrix_delete_cuda(env->output[i]);
		}else
		{
			wtk_cudnn_matrix_delete(env->output[i]);
		}
	}
	wtk_free(env->output);
	wtk_free(env);
}



__global__ void wtk_cudnn_update_bias_window(float *c,float *b,float *w,int col)
{
	int i,j;
	int threads=blockDim.x;
	int idx=threadIdx.x;

	i=blockIdx.x*col+idx;
	c[i]=(c[i]+b[idx])*w[idx];
	//c[i]=(c[i]+b[j])*w[j];
	for(j=i+threads,i=idx+threads;i<col;j+=threads,i+=threads)
	{
		c[j]=(c[j]+b[i])*w[i];
	}
}

int wtk_cudnn_env_update_trans(wtk_cudnn_env_t *env)
{
	wtk_cudnn_trans_t *trans=env->gpu_cfg->trans;
	int thread=env->gpu_cfg->max_thread;

	wtk_cudnn_update_bias_window<<<env->input->row,thread>>>(env->input->v,trans->b->v,trans->w->v,env->input->col);
	return 0;
}

void wtk_cudnn_matrix_print_cuda(wtk_cudnn_matrix_t *m)
{
	float *v;
	int n;

	n=m->row*m->col*sizeof(float);
	v=(float*)wtk_malloc(n);
	hipMemcpy(v,m->v,n,hipMemcpyDeviceToHost);
	//print_float(v,m->row*m->col);
	print_float(v,1*m->col);
	wtk_free(v);
}


void print_int_cu(int *a,int n)
{
	int *v;

	v=(int*)wtk_malloc(n*sizeof(int));
	hipMemcpy(v,a,n*sizeof(int),hipMemcpyDeviceToHost);
	//print_float(v,m->row*m->col);
	print_int(v,n);
	wtk_free(v);
}


__global__ void wtk_cudnn_sigmoid_cuda(float *f,int n)
{
	int i=(blockIdx.x*blockDim.x+threadIdx.x);

	if(i<n)
	{
		f[i]=1.0/(1.0+expf(-f[i]));
	}
}

__global__ void wtk_cudnn_softmax_cuda(float *f,int row,int col)
{
	int j=blockIdx.x;
	int threads=blockDim.x;
	int idx=threadIdx.x;
	int i,ki;
	__shared__ float aux[THREAD_PER_BLOCK];
	float max,sum;

	//find max;
	f+=j*col;
	aux[idx]=f[idx];
	for(ki=idx+threads;ki<col;ki+=threads)
	{
		if(aux[idx]<f[ki])
		{
			aux[idx]=f[ki];
		}
	}
	__syncthreads();
	ki=threads;
	while(ki>1)
	{
		i=((1+ki)>>1);//divide by two;
		if(idx<i)
		{
			j=idx+i;
			if(j<ki && aux[idx]<aux[j])
			{
				aux[idx]=aux[j];
			}
		}
		__syncthreads();
		ki=i;
	}
	max=aux[0];
	//f[0]=max;
	//return;
	__syncthreads();
	aux[idx]=f[idx]=expf(f[idx]-max);
	for(ki=idx+threads;ki<col;ki+=threads)
	{
		aux[idx]+=f[ki]=expf(f[ki]-max);
	}
	//return;
	__syncthreads();
	ki=threads;
	while(ki>1)
	{
		i=((1+ki)>>1);//divide by two;
		if(idx<i)
		{
			j=idx+i;
			if(j<ki)
			{
				aux[idx]+=aux[j];
			}
		}
		__syncthreads();
		ki=i;
	}
	sum=1.0/aux[0];
	//f[0]=sum;
	//return;
	__syncthreads();
	for(ki=idx;ki<col;ki+=threads)
	{
		f[ki]=log(f[ki]*sum);
	}
}

void wtk_cudnn_softmax2(float *f,int n)
{
	float max,sum;
	float *p,*e;

	max=wtk_math_max(f,n);
	//wtk_debug("max=%f\n",max);
	sum=0;
	p=f;e=p+n;
	while(p<e)
	{
		*p=expf(*p-max);
		//wtk_debug("%f\n",*p);
		sum+=*p;
		++p;
	}
	sum=1.0f/sum;
	//wtk_debug("sum=%f\n",sum);
	p=f;e=p+n;
	while(p<e)
	{
		*(p)=log(*p*sum);
		++p;
	}
}

#define wtk_cudnn_blk_mul(f,pa2,pb2,blk_size) \
		switch(blk_size) \
		{ \
		case 2: \
			f+=pa2[0]*pb2[0]; \
			f+=pa2[1]*pb2[1]; \
			break; \
		case 4: \
			f+=pa2[0]*pb2[0]; \
			f+=pa2[1]*pb2[1]; \
			f+=pa2[2]*pb2[2]; \
			f+=pa2[3]*pb2[3]; \
			break; \
		case 8: \
			f+=pa2[0]*pb2[0]; \
			f+=pa2[1]*pb2[1]; \
			f+=pa2[2]*pb2[2]; \
			f+=pa2[3]*pb2[3]; \
			f+=pa2[4]*pb2[4]; \
			f+=pa2[5]*pb2[5]; \
			f+=pa2[6]*pb2[6]; \
			f+=pa2[7]*pb2[7]; \
			break; \
		case 16: \
			f+=pa2[0]*pb2[0]; \
			f+=pa2[1]*pb2[1]; \
			f+=pa2[2]*pb2[2]; \
			f+=pa2[3]*pb2[3]; \
			f+=pa2[4]*pb2[4]; \
			f+=pa2[5]*pb2[5]; \
			f+=pa2[6]*pb2[6]; \
			f+=pa2[7]*pb2[7]; \
			f+=pa2[8]*pb2[8]; \
			f+=pa2[9]*pb2[9]; \
			f+=pa2[10]*pb2[10]; \
			f+=pa2[11]*pb2[11]; \
			f+=pa2[12]*pb2[12]; \
			f+=pa2[13]*pb2[13]; \
			f+=pa2[14]*pb2[14]; \
			f+=pa2[15]*pb2[15]; \
			break; \
		case 32: \
			f+=pa2[0]*pb2[0]; \
			f+=pa2[1]*pb2[1]; \
			f+=pa2[2]*pb2[2]; \
			f+=pa2[3]*pb2[3]; \
			f+=pa2[4]*pb2[4]; \
			f+=pa2[5]*pb2[5]; \
			f+=pa2[6]*pb2[6]; \
			f+=pa2[7]*pb2[7]; \
			f+=pa2[8]*pb2[8]; \
			f+=pa2[9]*pb2[9]; \
			f+=pa2[10]*pb2[10]; \
			f+=pa2[11]*pb2[11]; \
			f+=pa2[12]*pb2[12]; \
			f+=pa2[13]*pb2[13]; \
			f+=pa2[14]*pb2[14]; \
			f+=pa2[15]*pb2[15]; \
			f+=pa2[16]*pb2[16]; \
			f+=pa2[17]*pb2[17]; \
			f+=pa2[18]*pb2[18]; \
			f+=pa2[19]*pb2[19]; \
			f+=pa2[20]*pb2[20]; \
			f+=pa2[21]*pb2[21]; \
			f+=pa2[22]*pb2[22]; \
			f+=pa2[23]*pb2[23]; \
			f+=pa2[24]*pb2[24]; \
			f+=pa2[25]*pb2[25]; \
			f+=pa2[26]*pb2[26]; \
			f+=pa2[27]*pb2[27]; \
			f+=pa2[28]*pb2[28]; \
			f+=pa2[29]*pb2[29]; \
			f+=pa2[30]*pb2[30]; \
			f+=pa2[31]*pb2[31]; \
			break; \
		default: \
			{ \
				int k; \
				for(k=0;k<blk_size;++k) \
				{ \
					f+=pa2[k]*pb2[k]; \
				} \
			} \
		}


__global__ void wtk_cudnn_update_layer_wb(float *output,float *input,float *w,float *b,int col,int row,int mrow)
{
	int n,ki,kj,i;
	float f;

	n=(blockIdx.x*blockDim.x+threadIdx.x);
	ki=n/row;
	if(ki>=mrow)
	{
		return;
	}
	input+=ki*col;
	kj=n-ki*row;
	w+=kj*col;
	f=0;
	for(i=0;i<col;++i)
	{
		f+=input[i]*w[i];
	}
	//f+=b[j];
	//output[i][j]=f+b[j];
	output[ki*row+kj]=f+b[kj];
}


__global__ void wtk_cudnn_mat_mult_cuda_bias2(float *c,float *a,float *b,int blk_size,int acol,int arow,int brow,float *bias)
{
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	float f;
	int i,j,step;
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    int ky,kx1;
    float *pa1,*pb1,*pa2,*pb2;

	step=acol*ty+tx;
	kx1=blk_size*by;
    ky=kx1+ty;
	i=acol*kx1+step;
    kx1=blk_size*bx;
    //kx=kx1+ty;
	j=acol*kx1+step;
	kx1+=tx;
	pa1=As[ty]+tx;
	pb1=Bs[ty]+tx;
	pa2=As[ty];
	pb2=Bs[tx];
	f=0;
	//if(ky<arow || kx<brow || ((acol/step)*step!=acol))
	if(acol%blk_size>0) //((acol/step)*step!=acol))
	{
		for(step=0;step<acol;step+=blk_size)
		{
			if(tx+step<acol)
			{
				*pa1=a[i+step];
				*pb1=b[j+step];
			}else
			{
				*pa1=0;
				*pb1=0;
			}
			__syncthreads();
			wtk_cudnn_blk_mul(f,pa2,pb2,blk_size)
			__syncthreads();
		}
	}else
	{
		for(step=0;step<acol;step+=blk_size)
		{
			*pa1=a[i+step];
			*pb1=b[j+step];
			__syncthreads();
			wtk_cudnn_blk_mul(f,pa2,pb2,blk_size)
			__syncthreads();
		}
		//c[brow*ky+blk_size*bx+tx]=f+bias[kx];
	}
	if(ky<arow && kx1<brow)
	{
		c[brow*ky+kx1]=f+bias[kx1];
	}
}

__global__ void wtk_cudnn_mat_mult_cuda_bias3(float *c,float *a,float *b,int blk_size,int acol,int arow,int brow,float *bias)
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int bx=blockIdx.x;
    int by=blockIdx.y;
    int ky,kx;
    int step;
    float f;
    float *pa1,*pb1;
    float *pa2,*pb2;
    int i,j;

    pa1=As[ty]+tx;
    pb1=Bs[ty]+tx;
    i=acol*(blk_size*by+ty)+tx;
    j=acol*(blk_size*bx+ty)+tx;
    pa2=As[ty];
    pb2=Bs[tx];
    f=0;
	if(acol%blk_size>0) //((acol/step)*step!=acol))
	{
		for(step=0;step<acol;step+=blk_size)
		{
			if(tx+step<acol)
			{
				*pa1=a[i+step];
				*pb1=b[j+step];
			}else
			{
				*pa1=0;
				*pb1=0;
			}
			__syncthreads();
			wtk_cudnn_blk_mul(f,pa2,pb2,blk_size)
			__syncthreads();
		}
	}else
	{
		for(step=0;step<acol;step+=blk_size)
		{
			*pa1=a[i+step];
			*pb1=b[j+step];
			__syncthreads();
			wtk_cudnn_blk_mul(f,pa2,pb2,blk_size)
			__syncthreads();
		}
	}
    ky=blk_size*by+ty;
    if(ky<arow)
    {
    	kx=blk_size*bx+tx;
    	if(kx<brow)
    	{
    		c[brow*ky+kx]=f+bias[kx];
    	}
    }
}

__global__ void wtk_cudnn_mat_mult_cuda_bias4(float *c,float *a,float *b,int blk_size,int acol,int arow,int brow,float *bias)
{
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int bx=blockIdx.x;
    int by=blockIdx.y;
    int ky,kx;

    float f;
    float *pa1,*pb1;
    float *pa2,*pb2;
    int i,j;
    int endi,endi2;

    pa1=As[ty]+tx;
    pb1=Bs[ty]+tx;
    ky=blk_size*by+ty;
    i=acol*ky+tx;
    endi=i+acol;
	//kx=blk_size*bx+tx;
    kx=blk_size*bx;
    j=acol*(kx+ty)+tx;
    kx+=tx;
    pa2=As[ty];
    pb2=Bs[tx];
    f=bias[kx];
	if(acol%blk_size>0) //((acol/step)*step!=acol))
	{
		endi2=endi-tx;
		for(;i<endi;i+=blk_size,j+=blk_size)
		{
			if(i<endi2)
			{
				*pa1=a[i];
				*pb1=b[j];
			}else
			{
				*pa1=0;
				*pb1=0;
			}
			__syncthreads();
			wtk_cudnn_blk_mul(f,pa2,pb2,blk_size)
			__syncthreads();
		}
	}else
	{
		for(;i<endi;i+=blk_size,j+=blk_size)
		{
			*pa1=a[i];
			*pb1=b[j];
			__syncthreads();
			wtk_cudnn_blk_mul(f,pa2,pb2,blk_size)
			__syncthreads();
		}
	}
    if(ky<arow && kx<brow)
    {
    	c[brow*ky+kx]=f;
    }
}

int wtk_cudnn_process_layer_cuda(wtk_cudnn_env_t *env,wtk_cudnn_layer_t *layer,wtk_cudnn_matrix_t *input,wtk_cudnn_matrix_t *output)
{
	int thread=env->gpu_cfg->max_thread;
	int nx,ny;
	dim3 blk,grid;
	int blk_size=env->cu_cfg->blk_size;
	int arow,brow;

	output->row=input->row;
	arow=input->row;
	brow=layer->w->row;
	blk.z=1;
	blk.x=blk_size;
	blk.y=blk_size;
	grid.z=1;
	grid.x=(brow-1)/blk_size+1;
	grid.y=(arow-1)/blk_size+1;
	//wtk_cudnn_update_layer_wb<<<ny,nx>>>(output->v,input->v,layer->w->v,layer->b->v,input->col,layer->w->row,input->row);
	//wtk_debug("col=%d row=%d/%d\n",input->col,arow,brow);
	//hipMemset(output->v,0,output->row*output->col*sizeof(float));
	wtk_cudnn_mat_mult_cuda_bias4<<<grid,blk>>>(output->v,input->v,layer->w->v,blk_size,input->col,arow,brow,layer->b->v);
	switch(layer->type)
	{
	case wtk_dnn_sigmoid:
		//wtk_debug("row=%d col=%d\n",wtk_matrix_rows(output),wtk_matrix_cols(output));
		//wtk_cudnn_sigmoid(pf3-w->row,w->row);
		nx=output->row*output->col;
		ny=(int)((nx-1)/thread+1);
		//wtk_debug("nx=%d ny=%d thread=%d\n",nx,ny,thread);
		wtk_cudnn_sigmoid_cuda<<<ny,thread>>>(output->v,nx);
		//print_float_cu(output->v,layer->w->row);
		break;
	case wtk_dnn_softmax:
		//wtk_cudnn_softmax(pf3-w->row,w->row);
		//nx=(int)((output->col-1)/thread+1);
		//print_float_cu(output->v,layer->w->row);
		if(!env->cu_cfg->use_linear_output)
		{
			wtk_cudnn_softmax_cuda<<<output->row,thread>>>(output->v,output->row,output->col);
		}
		//print_float_cu(output->v,layer->w->row);
		//exit(0);
		break;
	case wtk_dnn_linear:
		break;
	default:
        wtk_debug("layer->type not in list. %d\n", layer->type);
        break;
	}
	//exit(0);
	return 0;
}


int wtk_cudnn_env_process(wtk_cudnn_env_t *env,wtk_cudnn_matrix_t *input)
{
	int n;
	int ret;
	int i;
	wtk_cudnn_matrix_t *output;
	wtk_cudnn_gpu_cfg_t *cfg=env->gpu_cfg;

	n=input->row*input->col*sizeof(float);
	ret=hipMemcpy(env->input->v,input->v,n,hipMemcpyHostToDevice);
	if(ret!=0){goto end;}
	env->input->row=input->row;
	//wtk_debug("n=%d/%d n=%d\n",input->row,input->col,n);
	//print_float(input->v+input->col*13,20);
	//print_float_cu(env->input->v+input->col*13,20);
	ret=wtk_cudnn_env_update_trans(env);
	if(ret!=0){goto end;}
	//print_float_cu(env->input->v,10);//env->input->col);
	input=env->input;
	for(i=0;i<cfg->nlayer;++i)
	{
		output=env->output[i];
		ret=wtk_cudnn_process_layer_cuda(env,cfg->layer[i],input,output);
		if(ret!=0){goto end;}
		//wtk_debug("========= i=%d ==========\n",i)
		//print_float_cu(output->v,10);//output->col);
		//exit(0);
		input=output;
	}
	env->host_output->row=output->row;
	ret=hipMemcpy(env->host_output->v,output->v,output->row*output->col*sizeof(float),hipMemcpyDeviceToHost);
	if(ret!=0){goto end;}
	//print_float(env->host_output->v,10);
	//exit(0);
	ret=0;
end:
	//exit(0);
	return ret;
}

void wtk_cudnn_test_cuda2()
{
#define N 30
	float pf[N];
	int i;
	float *dev_f;
	int ret;
	int nx;

	for(i=0;i<N;++i)
	{
		pf[i]=i;
	}
	ret=hipMalloc((void**)&(dev_f),N*sizeof(float));
	if(ret!=0){goto end;}
	ret=hipMemcpy(dev_f,pf,N*sizeof(float),hipMemcpyHostToDevice);
	if(ret!=0){goto end;}
	//3*10
	//print_float(pf,N);
	nx=4;
	//nx=1;
	//step=(int)(10.0/nx+0.5);
	//wtk_debug("step=%d\n",step);
	wtk_cudnn_softmax_cuda<<<3,nx>>>(dev_f,1,10);
	//print_float_cu(dev_f,N);
	for(i=0;i<3;++i)
	{
		wtk_cudnn_softmax2(pf+i*10,10);
	}
	//exit(0);
	//print_float(pf,N);
end:
	//exit(0);
	return;
}

char* wtk_cuda_dup_data(void *a,int n)
{
	char *dev_a=NULL;
	int ret;

	ret=hipMalloc((void**)&(dev_a),n);
	if(ret!=0){goto end;}
	if(a)
	{
		ret=hipMemcpy(dev_a,a,n,hipMemcpyHostToDevice);
		if(ret!=0){goto end;}
	}else
	{
		ret=hipMemset(dev_a,0,n);
		if(ret!=0){goto end;}
	}
end:
	if(ret!=0)
	{
		if(dev_a)
		{
			hipFree(dev_a);
		}
		dev_a=NULL;
	}
	return dev_a;
}

char* wtk_cuda_dup_dev_data(void *a,int n)
{
	char *v;

	v=(char*)wtk_malloc(n);
	hipMemcpy(v,a,n,hipMemcpyDeviceToHost);
	return v;
}


void wtk_cudnn_mat_mult(float *c,float *a,float *b,int arow,int acol,int brow)
{
	int i,j,k;
	float *pa,*pb;
	float f;

	for(i=0;i<arow;++i)
	{
		pa=a+i*acol;
		for(j=0;j<brow;++j)
		{
			pb=b+j*acol;
			f=0;
			for(k=0;k<acol;++k)
			{
				f+=pa[k]*pb[k];
			}
			//c[i][j]=f;
			c[i*brow+j]=f;
		}
	}
}



